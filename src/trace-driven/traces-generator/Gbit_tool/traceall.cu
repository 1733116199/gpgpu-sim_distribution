#include "hip/hip_runtime.h"
/* Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* Author: Oreste Villa, ovilla@nvidia.com - 2018 */

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include <string>
#include <map>
#include <inttypes.h>
#include <unistd.h>
#include <vector>
#include <iostream>
#include <bitset>
#include <sys/stat.h>
#include <sstream>
#include <algorithm>
#include <iterator>
/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* for _cuda_safe and GET_VAR* macros */
#include "macros.h"

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;

/* opcode to id map and reverse map  */
std::map<std::string, int> opcode_to_id_map;
std::map<int, std::string> id_to_opcode_map;

/* kernel instruction counter, updated by the GPU */
static __managed__ uint64_t total_dynamic_instr_counter = 0;
static __managed__ uint64_t reported_dynamic_instr_counter = 0;
static __managed__ uint64_t dynamic_instr_limit = 0;
uint64_t dynamic_instr_limit_input = 0;  //0 means no limit

#define MAX_SRC 4
/* information collected in the instrumentation function */
typedef struct {
	int cta_id_x;
	int cta_id_y;
	int cta_id_z;
	int warpid_tb;
	int warpid_sm;
	int sm_id;
	int opcode_id;
	uint64_t addrs[32];
	uint32_t vpc;
	bool is_mem;
	int32_t GPRDst;
	int32_t GPRSrcs[MAX_SRC];
	int32_t numSrcs;
	int32_t width;
	uint32_t active_mask;

} mem_access_t;

/* Instrumentation function that we want to inject, please note the use of
 * 1. extern "C" __device__ __noinline__
 *    To prevent "dead"-code elimination by the compiler.
 * 2. NVBIT_EXPORT_FUNC(dev_func)
 *    To notify nvbit the name of the function we want to inject.
 *    This name must match exactly the function name.
 */
extern "C" __device__ __noinline__ void instrument_mem(int pred, int opcode_id, int32_t vpc,
		uint32_t reg_high,
		uint32_t reg_low,
		int32_t imm,
		int32_t srcReg1, int32_t srcReg2, int32_t desReg, int32_t width) {
	if (!pred) {
		return;
	}

	uint32_t active_mask = __ballot(1);
	const int laneid = get_laneid();
	const int first_laneid = __ffs(active_mask) - 1;

	if (dynamic_instr_limit && total_dynamic_instr_counter >= dynamic_instr_limit)
		if (first_laneid == laneid) {
			atomicAdd((unsigned long long*)&total_dynamic_instr_counter, 1);
			return;
		}

	mem_access_t ma;

	/* collect memory address information */
	int64_t base_addr = (((uint64_t)reg_high) << 32) | ((uint64_t)reg_low);
	uint64_t addr = base_addr + imm;
	for (int i = 0; i < 32; i++) {
		ma.addrs[i] = __shfl(addr, i);
	}

	int4 cta = get_ctaid();
	int uniqe_threadId = threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
	ma.warpid_tb = uniqe_threadId/32;

	ma.cta_id_x = cta.x;
	ma.cta_id_y = cta.y;
	ma.cta_id_z = cta.z;
	ma.warpid_sm = get_warpid();
	ma.opcode_id = opcode_id;
	ma.is_mem = true;
	ma.vpc = vpc;
	ma.width = width;
	ma.GPRDst = desReg;
	ma.GPRSrcs[0] = srcReg1;
	ma.GPRSrcs[1] = srcReg2;
	ma.GPRSrcs[2] = -1;
	ma.GPRSrcs[3] = -1;
	ma.numSrcs = 2;
	ma.active_mask = active_mask;
	ma.sm_id =  get_smid();

	/* first active lane pushes information on the channel */
	if (first_laneid == laneid) {
		channel_dev.push(&ma, sizeof(mem_access_t));
		atomicAdd((unsigned long long*)&total_dynamic_instr_counter, 1);
		atomicAdd((unsigned long long*)&reported_dynamic_instr_counter, 1);
	}
}
NVBIT_EXPORT_FUNC(instrument_mem);


extern "C" __device__ __noinline__ void instrument_inst(int pred, int opcode_id,
		uint32_t vpc, int desReg, int srcReg1, int srcReg2, int srcReg3, int srcReg4, int srcNum) {
	if (!pred) {
		return;
	}

	int active_mask = __ballot(1);
	const int laneid = get_laneid();
	const int first_laneid = __ffs(active_mask) - 1;

	if (dynamic_instr_limit && total_dynamic_instr_counter >= dynamic_instr_limit)
		if (first_laneid == laneid) {
			atomicAdd((unsigned long long*)&total_dynamic_instr_counter, 1);
			return;
		}


	mem_access_t ma;

	int4 cta = get_ctaid();
	int uniqe_threadId = threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
	ma.warpid_tb = uniqe_threadId/32;

	ma.cta_id_x = cta.x;
	ma.cta_id_y = cta.y;
	ma.cta_id_z = cta.z;
	ma.warpid_sm = get_warpid();
	ma.opcode_id = opcode_id;
	ma.is_mem = false;
	ma.vpc = vpc;

	ma.GPRDst = desReg;
	ma.numSrcs = srcNum;    //this is the total src number including the register and others
	ma.GPRSrcs[0] = srcReg1;
	ma.GPRSrcs[1] = srcReg2;
	ma.GPRSrcs[2] = srcReg3;
	ma.GPRSrcs[3] = srcReg4;

	ma.active_mask = active_mask;
	ma.sm_id =  get_smid();

	/* first active lane pushes information on the channel */
	if (first_laneid == laneid) {
		channel_dev.push(&ma, sizeof(mem_access_t));
		atomicAdd((unsigned long long*)&total_dynamic_instr_counter, 1);
		atomicAdd((unsigned long long*)&reported_dynamic_instr_counter, 1);
	}
}

NVBIT_EXPORT_FUNC(instrument_inst);

void nvbit_at_init() {
	setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
	GET_VAR_INT(
			instr_begin_interval, "INSTR_BEGIN", 0,
			"Beginning of the instruction interval where to apply instrumentation");
	GET_VAR_INT(
			instr_end_interval, "INSTR_END", UINT32_MAX,
			"End of the instruction interval where to apply instrumentation");
	GET_VAR_LONG(
			dynamic_instr_limit_input, "DYNAMIC_INSTR_LIMIT", 0,
			"Limit of the number instructions to be printed, 0 means no limit");
	GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
	std::string pad(100, '-');
	printf("%s\n", pad.c_str());
}

/* instrument each memory instruction adding a call to the above instrumentation
 * function */
void nvbit_at_function_first_load(hipCtx_t ctx, hipFunction_t f) {

	dynamic_instr_limit = dynamic_instr_limit_input;

	const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
	if (verbose) {
		printf("Inspecting function %s at address 0x%lx\n",
				nvbit_get_func_name(ctx, f), nvbit_get_func_addr(f));
	}

	uint32_t cnt = 0;
	/* iterate on all the static instructions in the function */
	for (auto instr : instrs) {
		if (cnt < instr_begin_interval || cnt >= instr_end_interval ) {
			cnt++;
			continue;
		}
		//if (verbose) {
		instr->printDecoded();
		//}

		if (opcode_to_id_map.find(instr->getOpcode()) ==
				opcode_to_id_map.end()) {
			int opcode_id = opcode_to_id_map.size();
			opcode_to_id_map[instr->getOpcode()] = opcode_id;
			id_to_opcode_map[opcode_id] = instr->getOpcode();
		}

		int opcode_id = opcode_to_id_map[instr->getOpcode()];

		//TO DO: handle generic and TEX memory space
		if(instr->isLoad() && !instr->isStore() && instr->getMemOpType() != Instr::CONSTANT) {   //Mem load inst //ignore constant for now
			assert(instr->getNumOperands() == 2);

			/* get the operand */
			const Instr::operand_t *dst = instr->getOperand(0);
			const Instr::operand_t *src = instr->getOperand(1);

			assert(dst->type == Instr::REG);
			assert(src->type == Instr::MREF);

			/* insert call to the instrumentation function with its
			 * arguments */
			nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
			nvbit_add_call_arg_pred_val(instr);
			nvbit_add_call_arg_const_val32(instr, opcode_id);
			nvbit_add_call_arg_const_val32(instr, (int)instr->getOffset());
			if (instr->isExtended()) {
				nvbit_add_call_arg_reg_val(instr, (int)src->value[0] + 1);
			} else {
				nvbit_add_call_arg_reg_val(instr, (int)Instr::RZ);
			}
			nvbit_add_call_arg_reg_val(instr, (int)src->value[0]);
			nvbit_add_call_arg_const_val32(instr, (int)src->value[1]);
			nvbit_add_call_arg_const_val32(instr, (int)src->value[0]);
			nvbit_add_call_arg_const_val32(instr, -1);
			nvbit_add_call_arg_const_val32(instr, (int)dst->value[0]);
			nvbit_add_call_arg_const_val32(instr, (int)instr->getSize());
		}
		else if(instr->isStore() && !instr->isLoad() && instr->getMemOpType() != Instr::CONSTANT) {   //Mem store inst //ignore constant for now
			assert(instr->getNumOperands() == 2);

			/* get the operand */
			const Instr::operand_t *dst = instr->getOperand(0);
			const Instr::operand_t *src = instr->getOperand(1);

			assert(dst->type == Instr::MREF);
			assert(src->type == Instr::REG);

			/* insert call to the instrumentation function with its
			 * arguments */
			nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
			nvbit_add_call_arg_pred_val(instr);
			nvbit_add_call_arg_const_val32(instr, opcode_id);
			nvbit_add_call_arg_const_val32(instr, (int)instr->getOffset());
			if (instr->isExtended()) {
				nvbit_add_call_arg_reg_val(instr, (int)dst->value[0] + 1);
			} else {
				nvbit_add_call_arg_reg_val(instr, (int)Instr::RZ);
			}
			nvbit_add_call_arg_reg_val(instr, (int)dst->value[0]);
			nvbit_add_call_arg_const_val32(instr, (int)dst->value[1]);
			nvbit_add_call_arg_const_val32(instr, (int)dst->value[0]);
			nvbit_add_call_arg_const_val32(instr, (int)src->value[0]);
			nvbit_add_call_arg_const_val32(instr, -1);
			nvbit_add_call_arg_const_val32(instr, (int)instr->getSize());
		}
		else if(instr->isLoad() && instr->isStore() && instr->getMemOpType() != Instr::CONSTANT) {   //if it is load and store i.e. atomic inst 
			assert(instr->getNumOperands() == 2);

			/* get the operand */
			const Instr::operand_t *dst = instr->getOperand(0);
			const Instr::operand_t *src = instr->getOperand(1);

			assert(dst->type == Instr::MREF);
			assert(src->type == Instr::REG);

			/* insert call to the instrumentation function with its
			 * arguments */
			nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
			nvbit_add_call_arg_pred_val(instr);
			nvbit_add_call_arg_const_val32(instr, opcode_id);
			nvbit_add_call_arg_const_val32(instr, (int)instr->getOffset());
			if (instr->isExtended()) {
				nvbit_add_call_arg_reg_val(instr, (int)dst->value[0] + 1);
			} else {
				nvbit_add_call_arg_reg_val(instr, (int)Instr::RZ);
			}
			nvbit_add_call_arg_reg_val(instr, (int)dst->value[0]);
			nvbit_add_call_arg_const_val32(instr, (int)dst->value[1]);
			nvbit_add_call_arg_const_val32(instr, (int)dst->value[0]);
			nvbit_add_call_arg_const_val32(instr, (int)src->value[0]);
			nvbit_add_call_arg_const_val32(instr, -1);
			nvbit_add_call_arg_const_val32(instr, (int)instr->getSize());
		}
		else    //Other ALU, FP, DP insts
		{

			nvbit_insert_call(instr, "instrument_inst", IPOINT_BEFORE);
			nvbit_add_call_arg_pred_val(instr);
			nvbit_add_call_arg_const_val32(instr, opcode_id);
			nvbit_add_call_arg_const_val32(instr, (int)instr->getOffset());
			int srcNum = 0;
			for (int i = 0; i < MAX_SRC+1; i++) {
				/* get the operand "i" */
				if(i < instr->getNumOperands()) {
					const Instr::operand_t *op = instr->getOperand(i);
					if (op->type == Instr::REG)
						nvbit_add_call_arg_const_val32(instr, (int)op->value[0]);
					else
						nvbit_add_call_arg_const_val32(instr, -1);

					srcNum++;
				}
				else
					nvbit_add_call_arg_const_val32(instr, -1);
			}
			nvbit_add_call_arg_const_val32(instr, srcNum);
		}
		cnt++;
	}
}

__global__ void flush_channel() {
	/* push memory access with negative cta id to communicate the kernel is
	 * completed */
	mem_access_t ma;
	ma.cta_id_x = -1;
	channel_dev.push(&ma, sizeof(mem_access_t));

	/* flush channel */
	channel_dev.flush();
}

static FILE *resultsFile = NULL;
static FILE *kernelsFile= NULL;
static FILE *statsFile= NULL;
static int kernelid = 1;

unsigned old_total_insts = 0;
unsigned old_total_reported_insts = 0;


void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
		const char *name, void *params, hipError_t *pStatus) {
	if (skip_flag) return;

	if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
			cbid == API_CUDA_cuLaunchKernel) {
		cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

		if (!is_exit) {


			if (mkdir("traces", S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH) == -1) {
				if( errno == EEXIST ) {
					// alredy exists
				} else {
					// something else
					std::cout << "cannot create folder error:" << strerror(errno) << std::endl;
					return;
				}
			}

			int nregs;
			_cuda_safe(
					hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

			int shmem_static_nbytes;
			_cuda_safe(hipFuncGetAttribute(&shmem_static_nbytes,
					HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES,
					p->f));



			std::string func_name(nvbit_get_func_name(ctx, p->f));
			std::string::size_type end_pos = func_name.find('(');
			if (end_pos != std::string::npos)
			{
				// std::string::size_type pos = func_name.find('<');
				//if (pos != std::string::npos)
				//	end_pos = pos;

				//std::string::size_type start_pos = func_name.find(' ');
				//if (start_pos == std::string::npos)
				//	start_pos = 0;
				//else
				//	start_pos++;

				func_name = func_name.substr(0, end_pos);
			}

			char buffer[1024];
			sprintf (buffer, "./traces/%d-%s.trace", kernelid, func_name.c_str());

			resultsFile = fopen(buffer, "w");

			printf("Writing results to %s\n", buffer);

			fprintf(resultsFile, "-kernel name = %s",  nvbit_get_func_name(ctx, p->f));
			fprintf(resultsFile, "\n");
			fprintf(resultsFile, "-kernel id = %d",  kernelid);
			fprintf(resultsFile, "\n");
			fprintf(resultsFile, "-grid dim = (%d,%d,%d)",  p->gridDimX, p->gridDimY, p->gridDimZ);
			fprintf(resultsFile, "\n");
			fprintf(resultsFile, "-block dim = (%d,%d,%d)",  p->blockDimX, p->blockDimY, p->blockDimZ);
			fprintf(resultsFile, "\n");
			fprintf(resultsFile, "-shmem = %d",   shmem_static_nbytes + p->sharedMemBytes);
			fprintf(resultsFile, "\n");
			fprintf(resultsFile, "-nregs = %d",   nregs);
			fprintf(resultsFile, "\n");
			fprintf(resultsFile, "-cuda stream id = %d",  (uint64_t)p->hStream);
			fprintf(resultsFile, "\n\n");

			fprintf(resultsFile, "#traces format = threadblock_x threadblock_y threadblock_z warpid_tb sm_id warpid_sm PC mask dest_num reg_dests opcode src_num reg_srcs mem_width mem_addresses");
			fprintf(resultsFile, "\n");

			if (kernelid == 1) {
				kernelsFile = fopen("./traces/kernelslist", "w");
				statsFile = fopen("./traces/stats.csv", "w");
				fprintf(statsFile, "kernel name,total_insts,total_reported_insts\n");
			}
			else {
				kernelsFile = fopen("./traces/kernelslist", "a");
				statsFile = fopen("./traces/stats.csv", "a");
			}

			sprintf (buffer, "%d-%s.trace", kernelid, func_name.c_str());
			fprintf(kernelsFile, buffer);
			fprintf(kernelsFile, "\n");
			fclose(kernelsFile);

			fprintf(statsFile, buffer);
			fprintf(statsFile, ",");

			kernelid++;
			recv_thread_receiving = true;

		} else {
			/* make sure current kernel is completed */
			hipDeviceSynchronize();
			assert(hipGetLastError() == hipSuccess);

			/* make sure we prevent re-entry on the nvbit_callback when issuing
			 * the flush_channel kernel */
			skip_flag = true;

			/* issue flush of channel so we are sure all the memory accesses
			 * have been pushed */
			flush_channel<<<1, 1>>>();
			hipDeviceSynchronize();
			assert(hipGetLastError() == hipSuccess);

			/* unset the skip flag */
			skip_flag = false;

			/* wait here until the receiving thread has not finished with the
			 * current kernel */
			while (recv_thread_receiving) {
				pthread_yield();
			}

			unsigned total_insts_per_kernel =  total_dynamic_instr_counter - old_total_insts;
			old_total_insts = total_dynamic_instr_counter;

			unsigned reported_insts_per_kernel =  reported_dynamic_instr_counter - old_total_reported_insts;
			old_total_reported_insts = reported_dynamic_instr_counter;

			fprintf(statsFile, "");
			fprintf(statsFile, "%d,%d",total_insts_per_kernel,reported_insts_per_kernel);
			fprintf(statsFile, "\n");


			fclose(resultsFile);
			fclose(statsFile);
		}
	}
}

bool is_number(const std::string& s)
{
	std::string::const_iterator it = s.begin();
	while (it != s.end() && std::isdigit(*it)) ++it;
	return !s.empty() && it == s.end();
}

void *recv_thread_fun(void *) {
	char *recv_buffer = (char *)malloc(CHANNEL_SIZE);

	while (recv_thread_started) {
		uint32_t num_recv_bytes = 0;
		if (recv_thread_receiving &&
				(num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) >
		0) {
			uint32_t num_processed_bytes = 0;
			while (num_processed_bytes < num_recv_bytes) {
				mem_access_t *ma =
						(mem_access_t *)&recv_buffer[num_processed_bytes];

				/* when we get this cta_id_x it means the kernel has completed
				 */
				if (ma->cta_id_x == -1) {
					recv_thread_receiving = false;
					break;
				}

				fprintf(resultsFile, "%d ", ma->cta_id_x);
				fprintf(resultsFile, "%d ", ma->cta_id_y);
				fprintf(resultsFile, "%d ", ma->cta_id_z);
				fprintf(resultsFile, "%d ", ma->warpid_tb);
				fprintf(resultsFile, "%d ", ma->sm_id);
				fprintf(resultsFile, "%d ", ma->warpid_sm);
				fprintf(resultsFile, "0x%016lx ", ma->vpc); // Print the virtual PC.
				fprintf(resultsFile, "%-8.8" PRIx32 " ", ma->active_mask);
				if(ma->GPRDst >= 0) {
					fprintf(resultsFile, "1 ");
					fprintf(resultsFile, "R%d ", ma->GPRDst);
				}
				else
					fprintf(resultsFile, "0 ");

					// Print the opcode.
					fprintf(resultsFile, "%s ", id_to_opcode_map[ma->opcode_id].c_str());
				unsigned src_count=0;
				for (int s = 0; s < MAX_SRC; s++)      // GPR srcs count.
					if(ma->GPRSrcs[s] >= 0)  src_count++;
				fprintf(resultsFile, "%d ", src_count);

				for (int s = 0; s < MAX_SRC; s++)      // GPR srcs.
					if(ma->GPRSrcs[s] >= 0)  fprintf(resultsFile, "R%d ", ma->GPRSrcs[s]);

				//print addresses
				std::bitset<32> mask(ma->active_mask);
				if(ma->is_mem) {
					//fprintf(resultsFile, "%d ", ma->width);
					std::istringstream iss(id_to_opcode_map[ma->opcode_id]);
					std::vector<std::string> tokens;
					std::string token;
					while (std::getline(iss, token, '.')) {
						if (!token.empty())
							tokens.push_back(token);
					}
					if (tokens.size()>=3){
						if (is_number(tokens[2])){
							fprintf(resultsFile, "%d ", (std::stoi(tokens[2],nullptr)/8));
						}
						else{
							fprintf(resultsFile, "%d ", 4);
						}
					}
					else{
						fprintf(resultsFile, "%d ", 4);
					}

					for (int s = 0; s < 32; s++)
						if(mask.test(s))
							fprintf(resultsFile, "0x%016lx ", ma->addrs[s]);
				}
				else
				{
					fprintf(resultsFile, "0 ");
				}

				fprintf(resultsFile, "\n");

				num_processed_bytes += sizeof(mem_access_t);
			}
		}
	}
	free(recv_buffer);
	return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
	recv_thread_started = true;
	channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
	pthread_create(&recv_thread, NULL, recv_thread_fun, NULL);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
	if (recv_thread_started) {
		recv_thread_started = false;
		pthread_join(recv_thread, NULL);
	}
}
