#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, int *x, int *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  int *x, *y, *d_x, *d_y;
  x = (int*)malloc(N*sizeof(int));
  y = (int*)malloc(N*sizeof(int));

  hipMalloc(&d_x, N*sizeof(int)); 
  hipMalloc(&d_y, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    x[i] = 1;
    y[i] = 2;
  }

  hipMemcpy(d_x, x, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}
