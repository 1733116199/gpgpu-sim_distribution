#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TEXSIZE 4096
texture<float4, 1, hipReadModeElementType> texData;

void cuda_errcheck(const char *msg) {
  hipError_t err;
  if ((err = hipGetLastError()) != hipSuccess) {
    char errmsg[1024];
    sprintf(errmsg,"CUDA error %s: %s", msg, hipGetErrorString(err));
  }
}

void cuda_bind_texture_data(const float4 *t) {
    static hipArray *ct;
    if ( ! ct ) {
      hipMallocArray(&ct, &texData.channelDesc, TEXSIZE, 1);
      cuda_errcheck("allocating texDataArray");
    }     hipMemcpyToArray(ct, 0, 0, t, TEXSIZE*sizeof(float4), hipMemcpyHostToDevice); 
    cuda_errcheck("memcpy to texDataArray");

    texData.normalized = true;
    texData.addressMode[0] = hipAddressModeClamp;
    texData.addressMode[1] = hipAddressModeClamp;
    texData.filterMode = hipFilterModeLinear;

    hipBindTextureToArray(texData, ct);
    cuda_errcheck("binding texDataArray to texture");
}


__global__ void testKernel(float4 *data, float *coord)
{
    int gsize = blockDim.x * gridDim.x;
    int gid = threadIdx.x + blockDim.x * blockIdx.x;

    float norm_coord = (float)(gid + 32) / gsize;

    data[gid] = tex1D(texData, norm_coord);
    coord[gid] = norm_coord;
}

#define NUMTHREADS (TEXSIZE * 4)

int main(int argc, char *argv[])
{
    float4 h_texData[TEXSIZE];

    for (int t = 0; t < TEXSIZE; t++) {
        h_texData[t].x = t + 10000;
        h_texData[t].y = t + 20000;
        h_texData[t].z = t + 30000;
        h_texData[t].w = t + 40000;
    }

    float4 *h_output = new float4[NUMTHREADS];
    float4 *d_output;
    hipMalloc(&d_output, NUMTHREADS * sizeof(float4));
    cuda_errcheck("output malloc");

    float *h_coord = new float[NUMTHREADS];
    float *d_coord;
    hipMalloc(&d_coord, NUMTHREADS * sizeof(float));
    cuda_errcheck("coord malloc");

    hipSetDevice(0);
    cuda_errcheck("device init");

    cuda_bind_texture_data(h_texData);

    dim3 dimBlock(256, 1, 1);
    dim3 dimGrid(NUMTHREADS / 256, 1, 1);

    testKernel<<<dimGrid, dimBlock>>> (d_output, d_coord);
    cuda_errcheck("kernel launch");

    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, NUMTHREADS * sizeof(float4), hipMemcpyDeviceToHost);
    cuda_errcheck("output copy");

    hipMemcpy(h_coord, d_coord, NUMTHREADS * sizeof(float), hipMemcpyDeviceToHost);
    cuda_errcheck("coord copy");

    for (int t = 0; t < NUMTHREADS; t++) {
        printf("output[%d] (%.06f) = (%5.3f, %5.3f, %5.3f, %5.3f)\n", t, h_coord[t], 
            h_output[t].x, h_output[t].y, h_output[t].z, h_output[t].w);
    }

    return 0;
}

