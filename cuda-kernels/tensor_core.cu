/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}




#include <mma.h>
using namespace nvcuda;

// Must be multiples of 16 for wmma code to work
#define MATRIX_M (16)
#define MATRIX_N (16)
#define MATRIX_K (16)



// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;


// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16. 
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.
__global__ void wmma_example(half *a, half *b, float *c, int M, int N, int K, float alpha, float beta) {
   unsigned int start_time=0,end_time=0;
   // Leading dimensions. Packed with no transpositions.
   start_time=clock();
   int lda = M;
   int ldb = K;
   int ldc = M;

   // Tile using a 2D grid/
   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
 
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

   wmma::fill_fragment(c_frag, 0.0f);
   
   int i=0;
   int aRow = warpM * WMMA_M;
   int bCol = warpN * WMMA_N;
   int aCol = i;
   int bRow = i;


   // Bounds checking
   if (aRow < M && aCol < K && bRow < K && bCol < N) {
      wmma::load_matrix_sync(a_frag, a+aRow+aCol*lda, lda);
      wmma::load_matrix_sync(b_frag, b+bRow*ldb+bCol, ldb);
      wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
      //wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
   }
   int cRow = warpM * WMMA_M;
   int cCol = warpN * WMMA_N;
   wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
   end_time=clock();
   printf("clock=%d",end_time-start_time);
}

__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}

int main(int argc, char* argv[]) {
   float *a_fp32;
   float *b_fp32;
   half *a_fp16;
   half *b_fp16;

   float *c;
   float *c_cublas;
   float *c_wmma;

   float *c_host_cublas;
   float *c_host_wmma;
   float *a_host_wmma;
   float *b_host_wmma;
   float *c_init_host_wmma;
   
   
   hipEvent_t startWMMA;
   hipEvent_t stopWMMA;
   
   
   cudaErrCheck(hipEventCreate(&startWMMA));
   cudaErrCheck(hipEventCreate(&stopWMMA));
   
   
   
   
   // Use tensor cores
  
 
   cudaErrCheck(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

   cudaErrCheck(hipMalloc((void**)&c, MATRIX_M * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_wmma, MATRIX_M * MATRIX_N * sizeof(float)));

   c_host_wmma      = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   c_init_host_wmma = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   a_host_wmma      = (float*)malloc(MATRIX_M * MATRIX_K * sizeof(float));
   b_host_wmma      = (float*)malloc(MATRIX_K * MATRIX_N * sizeof(float));


   
//   printf("a_fp32\n");
   for(int m=0;m<MATRIX_M;m++){
	for(int n=0;n<MATRIX_K;n++){
		a_host_wmma[m*MATRIX_K+n]=(m*MATRIX_K+n)%10;
	}
	//printf(";\n");
   }
  // printf("b_fp32\n");
   for(int m=0;m<MATRIX_K;m++){
	for(int n=0;n<MATRIX_N;n++){
		b_host_wmma[m*MATRIX_N+n]=(m*MATRIX_N+n)%10;
//		printf("%f ",b_host_wmma[m*MATRIX_N+n]);
	}
//	printf(";\n");
   }
   cudaErrCheck(hipMemcpy(a_fp32,a_host_wmma,  MATRIX_M * MATRIX_K * sizeof(float), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(b_fp32,b_host_wmma,  MATRIX_K * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));

   // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp16, a_fp32, MATRIX_M * MATRIX_K);
   convertFp32ToFp16 <<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (b_fp16, b_fp32, MATRIX_K * MATRIX_N);

   for(int m=0;m<MATRIX_M;m++){
	for(int n=0;n<MATRIX_N;n++){
		c_init_host_wmma[m*MATRIX_N+n]=(m*MATRIX_M+n)%10;
	}
   }
   cudaErrCheck(hipMemcpy(c, c_init_host_wmma,  MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(c_wmma, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));

   float alpha = 1.0f;
   float beta = 1.0f;


   printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   
   // First: using WMMA
   dim3 gridDim;
   dim3 blockDim;
 
   // blockDim.x must be a multple of warpSize
   // 128x4 means we have 16 warps and a block computes a 64x64 output tile
   blockDim.x = 128;
   blockDim.y = 4;

   gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
   gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);
   
   printf("Running with wmma...\n");
   cudaErrCheck(hipEventRecord(startWMMA));
   wmma_example <<< 1, 32>>> (a_fp16, b_fp16, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
  // wmma_example <<< gridDim, blockDim >>> (a_fp16, b_fp16, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   cudaErrCheck(hipEventRecord(stopWMMA));


   

   // Error checking
   printf("\nChecking results...\n");
   cudaErrCheck(hipMemcpy(c_host_wmma, c_wmma, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
  // printf("c_host\n");
  // for(int m=0;m<MATRIX_M;m++){
//	for(int n=0;n<MATRIX_N;n++){
//	printf("%f ",c_host_wmma[m*MATRIX_N+n]);
//	}
//	printf(";\n");
  // }
   
      float wmmaTime;
      cudaErrCheck(hipEventSynchronize(stopWMMA));
      cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWMMA, stopWMMA));
      printf("wmma took %fms\n", wmmaTime);
      //printf("Clock=%d",stopWMMA-startWMMA);
      printf("\nFor a faster code using wmma you should check out the cudaTensorCoreGemm sample in the CUDA Toolkit.\nThis code was written as a demo only!\n\n");
   
   
   cudaErrCheck(hipEventDestroy(startWMMA));
   cudaErrCheck(hipEventDestroy(stopWMMA));

   
   cudaErrCheck(hipFree(a_fp32));
   cudaErrCheck(hipFree(b_fp32));
   cudaErrCheck(hipFree(a_fp16));
   cudaErrCheck(hipFree(b_fp16));

   cudaErrCheck(hipFree(c));
   cudaErrCheck(hipFree(c_wmma));
   
   free(c_host_wmma);

   cudaErrCheck(hipDeviceReset());
   return 0;
}


