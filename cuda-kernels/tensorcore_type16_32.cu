
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}

#include <mma.h>
using namespace nvcuda;

// Must be multiples of 16 for wmma code to work
#define MATRIX_M (16)
#define MATRIX_N (16)
#define MATRIX_K (16)


// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

__global__ void wmma_example(half *a, half *b, float *c,half *d_fp16, int M, int N, int K) {
   //unsigned int start_time=0,end_time=0;
   //start_time=clock();

   // Declare the fragments
   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> d_frag;

   // Bounds checking
   wmma::load_matrix_sync(a_frag, a, K);
   wmma::load_matrix_sync(b_frag, b, K);
   wmma::load_matrix_sync(c_frag, c, N,wmma::mem_col_major);
   
//   for(int i=0; i < c_frag.num_elements; i++) {
////  			c_frag.x[i]=c_frag.x[i]+c_frag.x[i];
//     	float temp=c_frag.x[i];
//	printf("THREAD%d:%d: %f \n",threadIdx.x,i,temp );
//   }
   wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

   for(int i=0; i < c_frag.num_elements; i++) {
 	d_frag.x[i]=c_frag.x[i];
   }
   wmma::store_matrix_sync(d_fp16, d_frag, N, wmma::mem_col_major);
   //printf("clock=%d",end_time-start_time);
}

__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}
__global__ void convertFp16ToFp32 (float *out, half *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}

int main(int argc, char* argv[]) {
   float *a_fp32;
   float *b_fp32;
   float *c_fp32;
   float *d_fp32;

   half *a_fp16;
   half *b_fp16;
   half *c_fp16;
   half *d_fp16;
   
   float *a_host_wmma;
   float *b_host_wmma;
   float *c_host_wmma;
   float *d_host_wmma;
   float *d_cal_host_wmma;

   hipEvent_t startWMMA;
   hipEvent_t stopWMMA;
   
   
   cudaErrCheck(hipEventCreate(&startWMMA));
   cudaErrCheck(hipEventCreate(&stopWMMA));
   
   // Use tensor cores
   cudaErrCheck(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&d_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&c_fp16, MATRIX_K * MATRIX_N * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&d_fp16, MATRIX_K * MATRIX_N * sizeof(half)));


   a_host_wmma      = (float*)malloc(MATRIX_M * MATRIX_K * sizeof(float));
   b_host_wmma      = (float*)malloc(MATRIX_K * MATRIX_N * sizeof(float));
   c_host_wmma      = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   d_host_wmma      = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   d_cal_host_wmma      = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

   //printf("a_fp32\n");
   for(int m=0;m<MATRIX_M;m++){
	for(int n=0;n<MATRIX_K;n++){
		a_host_wmma[m*MATRIX_K+n]=(m*MATRIX_K+n)%10;
	//	printf("%f ",a_host_wmma[m*MATRIX_K+n]);
	}
	//printf(";\n");
   }
  
   //printf("b_fp32\n");
   for(int m=0;m<MATRIX_K;m++){
	for(int n=0;n<MATRIX_N;n++){
		b_host_wmma[m*MATRIX_N+n]=(m*MATRIX_N+n)%10;
	//	printf("%f ",b_host_wmma[m*MATRIX_N+n]);
	}
	//	printf(";\n");
   }
   
   //printf("c_fp32\n");
   for(int m=0;m<MATRIX_M;m++){
	for(int n=0;n<MATRIX_N;n++){
		c_host_wmma[m*MATRIX_N+n]=(m*MATRIX_N+n);
		d_cal_host_wmma[m*MATRIX_N+n]=0;
	//	printf("%f ",c_host_wmma[m*MATRIX_N+n]);
	}
   }
   for(int m=0;m<MATRIX_M;m++){
	for(int n=0;n<MATRIX_N;n++){
		for(int k=0;k<MATRIX_K;k++){
			d_cal_host_wmma[m*MATRIX_N+n]+=	a_host_wmma[m*MATRIX_K+k]*b_host_wmma[k*MATRIX_K+n];
		}
		d_cal_host_wmma[m*MATRIX_N+n]+=c_host_wmma[m*MATRIX_N+n];
	}
   }


   cudaErrCheck(hipMemcpy(a_fp32,a_host_wmma,  MATRIX_M * MATRIX_K * sizeof(float), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(b_fp32,b_host_wmma,  MATRIX_K * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(c_fp32,c_host_wmma,  MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));

   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp16, a_fp32, MATRIX_M * MATRIX_K);
   convertFp32ToFp16 <<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (b_fp16, b_fp32, MATRIX_K * MATRIX_N);

   printf("\nM = %d, N = %d, K = %d. \n", MATRIX_M, MATRIX_N, MATRIX_K);
   
   printf("Running with wmma...\n");
   cudaErrCheck(hipEventRecord(startWMMA));
   wmma_example <<< 1, 32>>> (a_fp16, b_fp16, c_fp32, d_fp16 , MATRIX_M, MATRIX_N, MATRIX_K);
   cudaErrCheck(hipEventRecord(stopWMMA));
   cudaErrCheck(hipEventSynchronize(stopWMMA));

   convertFp16ToFp32 <<< (MATRIX_M * MATRIX_N + 255) / 256, 256 >>> (d_fp32, d_fp16, MATRIX_K * MATRIX_N);
  // Error checking
   printf("\nChecking results...\n");
   cudaErrCheck(hipMemcpy(d_host_wmma, d_fp32, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   
   printf("Results verified: cublas and WMMA agree.\n\n");
   float wmmaTime;
   cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWMMA, stopWMMA));
   printf("wmma took %fms\n", wmmaTime);
   
   cudaErrCheck(hipEventDestroy(startWMMA));
   cudaErrCheck(hipEventDestroy(stopWMMA));
   int t=600000;
   while(t-->0);

   printf("D_WMMA\n");
   for(int m=0;m<MATRIX_M;m++){
	for(int n=0;n<MATRIX_N;n++){
		printf("%.2f,",d_host_wmma[m*MATRIX_N+n]);
	}
	printf("\n");
   }
   printf("Check the result by executing the kernel on volta\n"); 
   cudaErrCheck(hipFree(a_fp32));
   cudaErrCheck(hipFree(b_fp32));
   cudaErrCheck(hipFree(c_fp32));
   cudaErrCheck(hipFree(d_fp32));
   cudaErrCheck(hipFree(a_fp16));
   cudaErrCheck(hipFree(b_fp16));
   cudaErrCheck(hipFree(c_fp16));
   cudaErrCheck(hipFree(d_fp16));

   free(a_host_wmma);
   free(b_host_wmma);
   free(c_host_wmma);
   free(d_host_wmma);
   cudaErrCheck(hipDeviceReset());
   return 0;
}


